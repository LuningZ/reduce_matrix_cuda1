/**
 * @file float.cu
 * @brief Parallel implementation (in the CPU and GPU) to reduce a matrix to a single value
 * 		Code for Task2 of MAP55616 Assignment1.
 * @author Luning
 * @version 1.0
 * @date 2023-03-17
 */


#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>
#include<math.h>
#include<string.h>
#include<stdbool.h>
#include<sys/time.h>

#define BLOCK_SIZE 256

float* init_matrix(int n, int m);
float* row_abs_sum(float* matrix, int n, int m);
float* col_abs_sum(float* matrix, int n, int m);
float vectorReduce(float* vec, int size);
double get_time(void);

/*
 * Kernel function to adds together the absolute value of each element of each row.
 * (into a vector of size n)
 */
__global__ void row_abs_sum_kernel(float* matrix, float* result, int n ,int m){
	int row = blockIdx.x * blockDim.x + threadIdx.x;
	if(row<n){
		float sum = 0.0f;
		for(int j=0;j<m;j++){
			sum += fabs(matrix[row*m + j]);
		}
		result[row] = sum;
	}
}

/*
 * Kernel function to adds together the absolute value of each element of each column.
 * (into a vector of size m)
 */
__global__ void col_abs_sum_kernel(float* matrix, float *result, int n, int m){
	int col = blockIdx.x* blockDim.x + threadIdx.x;
	if(col<m){
		float sum = 0.0f;
		for(int i=0;i<n;i++){
			sum += fabs(matrix[i*m + col]);
		}
		result[col] = sum;
	}
}

/*
 * Kernel function to reduce a vector to a single value by adding its components
 */
__global__ void vectorReduce_kernel(float* vec, float* result, int n){
	extern __shared__ float sdata[];
	int tid = threadIdx.x;
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	sdata[tid] = (i<n) ? vec[i] : 0.0f;
	__syncthreads();
	for (int s= blockDim.x/2;s>0;s>>=1){
		if(tid<s)
			sdata[tid] += sdata[tid + s];
		__syncthreads();
	}
	if(tid==0)
		result[blockIdx.x] = sdata[0];
}

/*
 * Main function
 */
int main(int argc, char ** argv){
	int n=10;
	int m=10;
	bool useRandomSeed = false;
	bool displayTiming = false;
	bool allTime = false;
	double t;

	// Parse command line arguments
	for (int i=1; i<argc; i++){
		if(strcmp(argv[i],"-n") == 0 && i+1 < argc){
			n=atoi(argv[i+1]);
		}else if (strcmp(argv[i],"-m") == 0 && i+1 <argc){
			m=atoi(argv[i+1]);
		}else if (strcmp(argv[i],"-r") == 0 ){
			useRandomSeed = true;
		}else if (strcmp(argv[i],"-t") == 0){
			displayTiming = true;
		}else if (strcmp(argv[i],"-a") == 0){
			allTime = true;
		}
	}

	// Set random seed
	if(useRandomSeed){
		struct timeval myRandom;
		gettimeofday(&myRandom,NULL);
		srand48((int)(myRandom.tv_usec));
	}
	else{
		srand48(123456);
	}

	// Initialize the matrix
	float* matrix = init_matrix(n,m);
	
	/* Only CPU Calculation ==============================*/
	// Calculate row sums
	t = get_time();
	float* rowSums = row_abs_sum(matrix,n,m);
	double time_row = (get_time() - t) * 1000000;

	// Calculate column sums
	t = get_time();
	float* colSums = col_abs_sum(matrix,n,m);
	double time_col = (get_time() - t) * 1000000;

	// Reduce row sums to a single value
	t = get_time();
	float rowSum = vectorReduce(rowSums,n);
	double time_reduce_row = (get_time() - t) * 1000000;

	// Reduce column sums to a single value
	t = get_time();
	float colSum = vectorReduce(colSums,m);
	double time_reduce_col = (get_time() - t) * 1000000;

	// Print results and timing (Serial Version)
	printf("Matrix size: %dx%d\n",n,m);
	printf("Use random seed: %s\n", useRandomSeed ? "true" : "false");
	printf("Print Timing: %s\n", displayTiming ? "true" : "false");
	printf("\n");

	printf("The results for CPU calculation:\n");
	printf("Row sum: %f\n", rowSum);
	printf("Column sum: %f\n", colSum);

	if(displayTiming && !allTime){
		printf("Serial Version (CPU) Time:\n");
		printf("Row sum duration: %lf microseconds\n", time_row);	
		printf("Column sum duration: %lf microseconds\n", time_col);
		printf("Row reduce duration: %lf microseconds\n", time_reduce_row);
		printf("Column reduce duration: %lf microseconds\n", time_reduce_col);
		printf("\n");
	}


	/* CPU and GPU Calculation ====================================== */
	// Allocate memory on CPU and GPU
	float *rowSums_cpu = (float*)malloc(n*sizeof(float));
	float *colSums_cpu = (float*)malloc(m*sizeof(float));
	float rowSum_cpu, colSum_cpu;
	float *result_cpu = (float*)malloc(BLOCK_SIZE * sizeof(float));

	float *matrix_gpu, *rowSums_gpu, *colSums_gpu, *result_gpu;
	hipMalloc((void**)&matrix_gpu, n*m*sizeof(float));
	hipMalloc((void**)&rowSums_gpu, n*sizeof(float));
	hipMalloc((void**)&colSums_gpu, m*sizeof(float));
	hipMalloc((void **)&result_gpu, BLOCK_SIZE*sizeof(float));

	// Copy matrix to GPU
	hipMemcpy(matrix_gpu, matrix, n*m*sizeof(float), hipMemcpyHostToDevice);

	// Calculate row-wise absolute sums on GPU
	t=get_time();
	int num_blocks_row = (n + BLOCK_SIZE - 1) / BLOCK_SIZE;
	row_abs_sum_kernel<<<num_blocks_row, BLOCK_SIZE>>>(matrix_gpu, rowSums_gpu, n, m);
	hipMemcpy(rowSums_cpu, rowSums_gpu, n * sizeof(float), hipMemcpyDeviceToHost);
	double time_row_gpu= (get_time() - t) * 1000000;

	// Calculate column-wise absolute sums on GPU
	t=get_time();
	int num_blocks_col = (m + BLOCK_SIZE - 1) / BLOCK_SIZE;
	col_abs_sum_kernel<<<num_blocks_col, BLOCK_SIZE>>>(matrix_gpu, colSums_gpu, n, m);
	hipMemcpy(colSums_cpu, colSums_gpu, m * sizeof(float), hipMemcpyDeviceToHost);
	double time_col_gpu= (get_time() - t) * 1000000;

	// Reduce row sums on GPU
	t=get_time();
	int num_blocks_reduce_row = (n + BLOCK_SIZE -1) / BLOCK_SIZE;
	vectorReduce_kernel<<<num_blocks_reduce_row, BLOCK_SIZE, BLOCK_SIZE * sizeof(float)>>>(rowSums_gpu, result_gpu, n);
	hipMemcpy(result_cpu, result_gpu, num_blocks_reduce_row * sizeof(float), hipMemcpyDeviceToHost);
	rowSum_cpu = 0.0f;
	for (int i=0;i<num_blocks_reduce_row; i++){
		rowSum_cpu += result_cpu[i]; 
	}
	double time_reduce_row_gpu= (get_time() - t) * 1000000;

	// Reduce column sums on GPU
	t=get_time();
	int num_blocks_reduce_col = (m + BLOCK_SIZE -1) / BLOCK_SIZE;
	vectorReduce_kernel<<<num_blocks_reduce_col, BLOCK_SIZE, BLOCK_SIZE * sizeof(float)>>>(colSums_gpu, result_gpu, m);
	hipMemcpy(result_cpu, result_gpu, num_blocks_reduce_col * sizeof(float), hipMemcpyDeviceToHost);
	colSum_cpu = 0.0f;
	for (int i=0;i<num_blocks_reduce_col; i++){
		colSum_cpu += result_cpu[i];
	}
	double time_reduce_col_gpu= (get_time() - t) * 1000000;

	// Verify results
	printf("The results for CPU and GPU Calculation:\n");
	printf("Row sum: %f\n",rowSum_cpu);
	printf("Column sum: %f\n",colSum_cpu);

	// Print time and compare
	if(displayTiming && allTime){
		printf("\nTime Compare:\n");
		printf("Row sum duration:\nSerial Version: %lf microseconds ; Parallel Version: %lf microseconds\n", time_row, time_row_gpu);	
		printf("Column sum duration:\nSerial Version: %lf microseconds ; Parallel Version: %lf microseconds\n", time_col, time_col_gpu);
		printf("Row reduce duration:\nSerial Version: %lf microseconds ; Parallel Version: %lf microseconds\n", time_reduce_row, time_reduce_row_gpu);
		printf("Column reduce duration:\nSerial Version: %lf microseconds ; Parallel Version: %lf microseconds\n", time_reduce_col, time_reduce_col_gpu);
		printf("\n");
	}

	// Free device memory
	hipFree(matrix_gpu);
	hipFree(rowSums_gpu);
	hipFree(colSums_gpu);
	hipFree(result_gpu);

	free(matrix);
	free(rowSums);
	free(colSums);
	free(rowSums_cpu);
	free(colSums_cpu);
	free(result_cpu);

	return 0;
}

/*
 * Function to allocate and initialize the matrix.
 */
float* init_matrix(int n, int m){
	float* matrix = (float*)malloc(n * m * sizeof(float));
	for(int i=0;i<n*m;i++){
		matrix[i] = ((float)(drand48())*10.0)-5.0;
	}
	return matrix;
}

/*
 * Function to get the current time.
 */
double get_time(void){
	struct timeval tv;
	double t;

	gettimeofday(&tv, NULL);
	t= tv.tv_sec + (double)tv.tv_usec * 1e-6;

	return t;
}

/*
 * Function to adds together the absolute value of each element of each row.
 * (into a vector of size n)
 */
float* row_abs_sum(float* matrix, int n,int m){
	float* rowSums=(float*)malloc(n*sizeof(float));
	memset(rowSums,0, n*sizeof(float));
	for (int i=0;i<n;i++){
		for(int j=0;j<m;j++){
			rowSums[i] += fabs(matrix[i*m+j]);
		}
	}
	return rowSums;
}

/*
 * Function to adds together the absolute value of each element of each column.
 * (into a vector of size m)
 */
float* col_abs_sum(float* matrix, int n, int m){
	float* colSums=(float*)malloc(m*sizeof(float));
	memset(colSums,0, m*sizeof(float));
	for(int i=0;i<n;i++){
		for(int j=0;j<m;j++){
			colSums[j] += fabs(matrix[i*m+j]);
		}
	}
	return colSums;
}

/*
 * Function to reduce a vector to a single value by adding its components
 */
float vectorReduce(float* vec, int size){
	float sum = 0.0f;
	for (int i=0; i<size; i++){
		sum+=vec[i];
	}
	return sum;
}

